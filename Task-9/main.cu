#include "hip/hip_runtime.h"
//
//  main.cpp
//
//
//  Created by Elijah Afanasiev on 25.09.2018.
//
//

// System includes
#include <assert.h>
#include <stdio.h>
#include <chrono>
#include <cstdlib>
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void vectorAddGPU(float* a, float* b, float* c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

void unified_sample(int size = 1048576) {
  int n = size;
  int nBytes = n * sizeof(float);

  float *a, *b, *c;

  hipEvent_t eStart, eStop;
  hipEventCreate(&eStart);
  hipEventCreate(&eStop);

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  hipMallocManaged(&a, nBytes);
  hipMallocManaged(&b, nBytes);
  hipMallocManaged(&c, nBytes);

  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float)RAND_MAX;
    b[i] = rand() / (float)RAND_MAX;
  }

  hipEventRecord(eStart);
  vectorAddGPU<<<grid, block>>>(a, b, c, n);
  hipEventRecord(eStop);
  hipDeviceSynchronize();

  float eTime;
  hipEventElapsedTime(&eTime, eStart, eStop);
  std::cout << "Unified Memory copying runs for : " << eTime<< " ms."<<std::endl;

  hipDeviceSynchronize();
}

void pinned_sample(int size = 1048576) {
  int n = size;
  int nBytes = n * sizeof(float);

  float *h_a, *h_b, *h_c;
  float *d_a, *d_b, *d_c;

  hipEvent_t pStart, pStop;
  hipEventCreate(&pStart);
  hipEventCreate(&pStop);

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  hipHostMalloc(&h_a, nBytes);
  hipHostMalloc(&h_b, nBytes);
  hipHostMalloc(&h_c, nBytes);
  hipMalloc(&d_a, nBytes);
  hipMalloc(&d_b, nBytes);
  hipMalloc(&d_c, nBytes);

  for (int i = 0; i < n; i++) {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
    h_c[i] = 0;
  }

  hipEventRecord(pStart);
  hipMemcpy(d_a, h_a, nBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, nBytes, hipMemcpyHostToDevice);
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, n);

  hipEventRecord(pStop);
  hipDeviceSynchronize();

  float pTime;
  hipEventElapsedTime(&pTime, pStart, pStop);
  std::cout << "Pinned Memory copying runs for : " << pTime<< " ms.\n";

  hipDeviceSynchronize();
}

void usual_sample(int size = 1048576) {
  int n = size;

  int nBytes = n * sizeof(float);

  float *a, *b;  // host data
  float* c;      // results

  a = (float*)malloc(nBytes);
  b = (float*)malloc(nBytes);
  c = (float*)malloc(nBytes);

  float *a_d, *b_d, *c_d;

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  for (int i = 0; i < n; i++) {
    a[i] = rand() / (float)RAND_MAX;
    b[i] = rand() / (float)RAND_MAX;
    c[i] = 0;
  }

  printf("Allocating device memory on host..\n");

  hipMalloc((void**)&a_d, n * sizeof(float));
  hipMalloc((void**)&b_d, n * sizeof(float));
  hipMalloc((void**)&c_d, n * sizeof(float));

  printf("Copying to device..\n");

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  hipMemcpy(a_d, a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, n * sizeof(float), hipMemcpyHostToDevice);

  printf("Doing GPU Vector add\n");

  vectorAddGPU<<<grid, block>>>(a_d, b_d, c_d, n);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("time: %f ms\n", milliseconds);

  hipDeviceSynchronize();

  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);
  free(a);
  free(b);
  free(c);
}

int main(int argc, char** argv) {
  usual_sample(atoi(argv[1]));
  pinned_sample(atoi(argv[1]));
  unified_sample(atoi(argv[1]));
  return 0;
}