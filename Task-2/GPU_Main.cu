#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <iostream>
#include<sys/time.h>
using namespace std;
__global__ void transposeKernel(const double* A, double* AT, int N) {
  int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
  int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    AT[yIndex+xIndex*N]=A[xIndex+yIndex*N];
}
int main(void) {
    int rank=100;//for 50*50 mart
    struct timeval start, end;
    int N =rank;

    dim3 threadPerBlock(N, N);
    dim3 blockNumber((N+threadPerBlock.x-1)/ threadPerBlock.x, (N+threadPerBlock.y-1)/ threadPerBlock.y );

    size_t size = N * N * sizeof(double);

    double* h_A = (double*)malloc(size);

    double* h_AT = (double*)malloc(size);

    for (int i = 0; i < N * N; i++) {
      h_A[i] = i +1;
    }

    int i = 0, k = 0;
    gettimeofday(&start,NULL);
    
    while (i < N * N) {
      for (int j = k; j < N * N; j += N) {
        h_AT[i++] = h_A[j];
      }
      k++;
    }

    gettimeofday(&end,NULL);
    int timeuseCPU = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time for cpu is " << timeuseCPU<< "us" <<endl;

    double* d_A = NULL;
    double* d_AT = NULL;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_AT, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    gettimeofday(&start,NULL);
    transposeKernel<<<blockNumber, threadPerBlock>>>(d_A, d_AT, N);
    hipDeviceSynchronize();
    gettimeofday(&end,NULL);
    int timeuseGPU = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    cout << "total time use in GPU is " << timeuseGPU<< "us" <<endl;
    hipMemcpy(h_AT, d_AT, size, hipMemcpyDeviceToHost);
    if(timeuseGPU<timeuseCPU){
        cout<<"GPU is faster than CPU for "<<timeuseCPU-timeuseGPU<<" us"<<endl;
    }else{
        cout<<"CPU is faster than GPU for "<<timeuseGPU-timeuseCPU<<" us"<<endl;
    }
    free(h_A);
    free(h_AT);
    hipFree(d_A);
    hipFree(d_AT);
  return 0;
}