
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <sys/time.h>
#define STREAMS_NUM 8
__global__ void plus(float *a, float *b, float *c, int n, int offset) {

    int i = blockIdx.x*blockDim.x + threadIdx.x + offset;
        c[i] = a[i] + b[i];
}
int main(void){
    int n = 1024*1024;
    int size = n*sizeof(float);
    struct timeval start, end;
    float *a, *b;  
    float *c;

    hipHostAlloc( (void**) &a, size ,hipHostMallocDefault );
    hipHostAlloc( (void**) &b, size ,hipHostMallocDefault );
    hipHostAlloc( (void**) &c, size ,hipHostMallocDefault );

    float *a_d,*b_d,*c_d;

    for(int i=0; i < n; i++) {
        a[i] = 20.0;
        b[i] = 10.0;
    }
    hipMalloc((void **)&a_d,size);
    hipMalloc((void **)&b_d,size);
    hipMalloc((void **)&c_d,size);
    const int StreamSize = n / STREAMS_NUM;
    hipStream_t Stream[STREAMS_NUM];

    for (int i = 0; i < STREAMS_NUM; i++)
        hipStreamCreate(&Stream[i]);

    dim3 block(1024);
    dim3 grid((n- 1)/1024 + 1);
    gettimeofday( &start, NULL );
    for ( int i = 0; i < STREAMS_NUM; i++) {

        int Offset = i * StreamSize;

        hipMemcpyAsync(&a_d[Offset], &a[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[i]);
        hipMemcpyAsync(&b_d[Offset], &b[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[i]);
        hipMemcpyAsync(&c_d[Offset], &c[Offset], StreamSize * sizeof(float), hipMemcpyHostToDevice, Stream[i]);

        plus<<<grid, block>>>(a_d, b_d, c_d, StreamSize, Offset);

        hipMemcpyAsync(&a[Offset], &a_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[i]);
        hipMemcpyAsync(&b[Offset], &b_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[i]);
        hipMemcpyAsync(&c[Offset], &c_d[Offset], StreamSize * sizeof(float), hipMemcpyDeviceToHost, Stream[i]);
    }
    gettimeofday(&end,NULL);
    int timeuseGPU = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    std::cout<<"total time use in GPU-Stream is "<<timeuseGPU<<" us "<<std::endl;
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    hipHostFree(a);
    hipHostFree(b);
    hipHostFree(c);
}