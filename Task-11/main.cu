#include <assert.h>
#include <hip/hip_runtime.h>


#include <iostream>
#include <chrono>
#include <cstdlib>
#include <iostream>

__global__ void Plus(float* a, float* b, float* c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

void twoGPU(int size) {
  int n = size;
  int work_per_gpu = (n - 1) / 2 + 1;
  int nBytes = n * sizeof(float);
  int nBytes_per_gpu = work_per_gpu * sizeof(float);
  float *h_a, *h_b, *h_c;
  h_a = (float*)malloc(nBytes);
  h_b = (float*)malloc(nBytes);
  h_c = (float*)malloc(nBytes);
  hipHostRegister(h_a, nBytes, 0);
  hipHostRegister(h_b, nBytes, 0);
  hipHostRegister(h_c, nBytes, 0);

  for (int i = 0; i < n; i++) {
    h_a[i] = i;
    h_b[i] = i + 1;
  }

  float *d_a0, *d_b0, *d_c0;
  float *d_a1, *d_b1, *d_c1;

  hipSetDevice(0);
  hipMalloc(&d_a0, nBytes_per_gpu);
  hipMalloc(&d_b0, nBytes_per_gpu);
  hipMalloc(&d_c0, nBytes_per_gpu);
  hipSetDevice(1);
  hipMalloc(&d_a1, nBytes_per_gpu);
  hipMalloc(&d_b1, nBytes_per_gpu);
  hipMalloc(&d_c1, nBytes_per_gpu);
  hipSetDevice(0);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  const int BLOCK_SIZE = 1024;
  const int GRID_SIZE = (work_per_gpu - 1) / BLOCK_SIZE + 1;

  hipEventRecord(start);

  hipSetDevice(0);
  hipMemcpyAsync(d_a0, &h_a[0], nBytes_per_gpu, hipMemcpyHostToDevice);
  hipMemcpyAsync(d_b0, &h_b[0], nBytes_per_gpu, hipMemcpyHostToDevice);

  Plus<<<GRID_SIZE, BLOCK_SIZE>>>(d_a0, d_b0, d_c0, n);

  hipMemcpyAsync(&h_c[0], d_c0, nBytes_per_gpu, hipMemcpyDeviceToHost);

  hipSetDevice(1);
  hipMemcpyAsync(d_a1, &h_a[work_per_gpu], nBytes_per_gpu,
                  hipMemcpyHostToDevice);
  hipMemcpyAsync(d_b1, &h_b[work_per_gpu], nBytes_per_gpu,
                  hipMemcpyHostToDevice);
  Plus<<<GRID_SIZE, BLOCK_SIZE>>>(d_a1, d_b1, d_c1, n);

  hipMemcpyAsync(&h_c[work_per_gpu], d_c1, nBytes_per_gpu,
                  hipMemcpyDeviceToHost);

  hipDeviceSynchronize();
  hipSetDevice(0);
  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float twoGPU = 0;
  hipEventElapsedTime(&twoGPU, start, stop);
  std::cout<<"Two GPUs run for :"<<twoGPU<<" ms "<<std::endl;

  hipFree(d_a0);
  hipFree(d_b0);
  hipFree(d_c0);
  hipSetDevice(1);
  hipFree(d_a1);
  hipFree(d_b1);
  hipFree(d_c1);
  hipSetDevice(0);
  hipHostUnregister(h_a);
  hipHostUnregister(h_b);
  hipHostUnregister(h_c);
  free(h_a);
  free(h_b);
  free(h_c);
}

void oneGPU(int size) {
  int n = size;
  int nBytes = n * sizeof(float);

  float *h_a, *h_b, *h_c;

  h_a = (float*)malloc(nBytes);
  h_b = (float*)malloc(nBytes);
  h_c = (float*)malloc(nBytes);

  float *d_a, *d_b, *d_c;

  dim3 block(256);
  dim3 grid((unsigned int)ceil(n / (float)block.x));

  for (int i = 0; i < n; i++) {
    h_a[i] =20.0;
    h_b[i] = 10.0;
  }

  hipMalloc((void**)&d_a, n * sizeof(float));
  hipMalloc((void**)&d_b, n * sizeof(float));
  hipMalloc((void**)&d_c, n * sizeof(float));
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);
  Plus<<<grid, block>>>(d_a, d_b, d_c, n);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float sigTime = 0;
  hipEventElapsedTime(&sigTime, start, stop);
  std::cout<<"One GPU runs for :"<<sigTime<<" ms "<<std::endl;

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  free(h_a);
  free(h_b);
  free(h_c);
}

int main(int argc, char* argv[]) {
  assert(argc==2);
  oneGPU(atoi(argv[1]));
  twoGPU(atoi(argv[1]));
  return 0;
}
